#include "hip/hip_runtime.h"
#include <iostream>
#include <hipcub/hipcub.hpp>

template <int BLOCK_THREADS, typename T>
__global__ void ExampleKernel(const T* in, T* out){
    
    // Specialize the template for double precision and BLOCK_THREADS threads w/ 4 items per thread 
    typedef hipcub::BlockLoad<const T*, BLOCK_THREADS, 4> BlockLoadT;
    // Specialize the template for double precision and BLOCK_THREADS threads    
    typedef hipcub::BlockReduce<T, BLOCK_THREADS> BlockReduceT;
    // Declare shared storage
    
    __shared__ union {
        typename BlockLoadT::TempStorage load;
        typename BlockReduceT::TempStorage reduce;
    } temp_storage;
    
    T items[4];
    
    BlockLoadT(temp_storage.load).Load(in, items);
    __syncthreads();
    
    // Instantiate an instance of BlockReduceT
    T result = BlockReduceT(temp_storage.reduce).Sum(items);
    
    if (threadIdx.x == 0){
        *out = result;
    }
}


int main(){
    
    double* d = new double[4096];
    double* d_gpu = NULL;
    double result = 0.0;
    double* result_gpu = NULL; 
    
    for (int i = 0; i < 4096; ++i) {
        d[i] = 1.0/4096;
    }
    
    // Allocate memory on the GPU
    hipMalloc(&d_gpu, 4096 * sizeof(double));
    hipMalloc(&result_gpu, sizeof(double));
    hipMemcpy(d_gpu, d, 4096 * sizeof(double), hipMemcpyHostToDevice);
    // Call the kernel
    ExampleKernel<1024><<<1, 1024>>>(d_gpu, result_gpu);
    hipMemcpy(&result, result_gpu, sizeof(double), hipMemcpyDeviceToHost);
    std::cout << "The result is " << result << std::endl;
    
}
