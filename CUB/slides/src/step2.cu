#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>

__global__ void ExampleKernel(...){

    // Specialize the template for double precision and 128 threads    
    typedef hipcub::BlockReduce<double, 128> BlockReduceT;
    
    // Declare shared storage
    __shared__ typename BlockReduceT::TempStorage temp_storage;
    
    double items[4];
    
    ...
}