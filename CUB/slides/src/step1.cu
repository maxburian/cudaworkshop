#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>

__global__ void ExampleKernel(...){
    
    // Specialize the template for double precision and 128 threads
    typedef hipcub::BlockReduce<double, 128> BlockReduceT;
    
    ...
}