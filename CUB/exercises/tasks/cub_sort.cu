#include "hip/hip_runtime.h"
#include <iostream>
#include <hiprand.h>
#include <hipcub/hipcub.hpp>

template <typename T>
__global__ void sort(const T* data_in, T* data_out){
    
    // TODO: Specialize the sort and declare the temporary storage.
    typedef ...
    __shared__ ...
    
    
    T items[8];
    int i0 = 8 * (blockIdx.x * blockDim.x + threadIdx.x);
    for (int i = 0; i < 8; ++i){
        items[i] = data_in[i0 + i];
    }
    // TODO: Perform the sort
    
    for (int i = 0; i < 8; ++i){
        data_out[i0 + i] = items[i];
    }
}

int main(){
    double* d_gpu = NULL;
    double* result_gpu = NULL;
    double* data_sorted = new double[4096];
    // Allocate memory on the GPU
    hipMalloc(&d_gpu, 4096 * sizeof(double));
    hipMalloc(&result_gpu, 4096 * sizeof(double));
    
    hiprandGenerator_t gen;
    //     Create generator
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    //     Fill array with random numbers
    hiprandGenerateNormalDouble(gen, d_gpu, 4096, 0.0, 1.0);
    //     Destroy generator
    hiprandDestroyGenerator(gen);
    
    // Sort data    
    sort<<<1, 512>>>(d_gpu, result_gpu);
    
    hipMemcpy(data_sorted, result_gpu, 4096 * sizeof(double), hipMemcpyDeviceToHost);
    // Write the sorted data to standard out
    
    for (int i = 0; i < 4095; ++i){
        std::cout << data_sorted[i] << ", ";
    }
    
    std::cout << data_sorted[4095] << std::endl;
}
