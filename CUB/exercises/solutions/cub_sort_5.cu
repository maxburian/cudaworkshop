#include "hip/hip_runtime.h"
#include <iostream>
#include <hiprand.h>
#include <hipcub/hipcub.hpp>

template <int BLOCK_WIDTH, int ITEMS_PER_THREAD, 
          hipcub::BlockLoadAlgorithm BLOCK_LOAD_ALGO,
          hipcub::BlockStoreAlgorithm BLOCK_STORE_ALGO,
          typename T>
__global__ void sort(const T* data_in, T* data_out){
    
    typedef hipcub::BlockLoad<T, BLOCK_WIDTH, ITEMS_PER_THREAD, BLOCK_LOAD_ALGO> BlockLoadT;
    typedef hipcub::BlockRadixSort<T, BLOCK_WIDTH, ITEMS_PER_THREAD> BlockRadixSortT;
    typedef hipcub::BlockStore<T, BLOCK_WIDTH, ITEMS_PER_THREAD, BLOCK_STORE_ALGO> BlockStoreT;
    
    __shared__ union {
        typename BlockLoadT::TempStorage load;
        typename BlockRadixSortT::TempStorage sort;
        typename BlockStoreT::TempStorage store;
    } tmp_storage;
    
    
    T items[ITEMS_PER_THREAD];
    BlockLoadT(tmp_storage.load).Load(data_in + blockIdx.x * BLOCK_WIDTH * ITEMS_PER_THREAD, items);
    __syncthreads();

    BlockRadixSortT(tmp_storage.sort).Sort(items);
    __syncthreads();
    
    BlockStoreT(tmp_storage.store).Store(data_out + blockIdx.x * BLOCK_WIDTH * ITEMS_PER_THREAD, 
                items);
    
}

int main(){
    double* d_gpu = NULL;
    double* result_gpu = NULL;
    double* data_sorted = new double[1000*4096];
    // Allocate memory on the GPU
    hipMalloc(&d_gpu, 1000*4096 * sizeof(double));
    hipMalloc(&result_gpu, 1000*4096 * sizeof(double));
    
    hiprandGenerator_t gen;
    //     Create generator
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    //     Fill array with random numbers
    hiprandGenerateNormalDouble(gen, d_gpu, 1000*4096, 0.0, 1.0);
    //     Destroy generator
    hiprandDestroyGenerator(gen);
    
    // Sort data    
    sort<512, 8, hipcub::BLOCK_LOAD_TRANSPOSE, hipcub::BLOCK_STORE_TRANSPOSE><<<1000, 512>>>(d_gpu, 
                                                                                       result_gpu);
    
    sort<256, 16, hipcub::BLOCK_LOAD_DIRECT, hipcub::BLOCK_STORE_DIRECT><<<1000, 256>>>(d_gpu, 
                                                                                  result_gpu);
    hipMemcpy(data_sorted, result_gpu, 1000*4096*sizeof(double), hipMemcpyDeviceToHost);

    // Write the sorted data to standard out
    for (int i = 0; i < 4095; ++i){
        std::cout << data_sorted[i] << ", ";
    }
    
    std::cout << data_sorted[4095] << std::endl;
}
