#include "hip/hip_runtime.h"
#include <iostream>
#include <hiprand.h>
#include <hipcub/hipcub.hpp>

template <typename T>
__global__ void sort(const T* data_in, T* data_out){
    
    typedef hipcub::BlockRadixSort<T, 512, 8> BlockRadixSortT;
    __shared__ typename BlockRadixSortT::TempStorage tmp_sort;
    
    
    T items[8];
    int i0 = 8 * (blockIdx.x * blockDim.x + threadIdx.x);
    for (int i = 0; i < 8; ++i){
        items[i] = data_in[i0 + i];
    }
    
    BlockRadixSortT(tmp_sort).Sort(items);
    
    for (int i = 0; i < 8; ++i){
        data_out[i0 + i] = items[i];
    }
}

int main(){
    double* d_gpu = NULL;
    double* result_gpu = NULL;
    double* data_sorted = new double[4096];
    // Allocate memory on the GPU
    hipMalloc(&d_gpu, 4096 * sizeof(double));
    hipMalloc(&result_gpu, 4096 * sizeof(double));
    
    hiprandGenerator_t gen;
    //     Create generator
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    //     Fill array with random numbers
    hiprandGenerateNormalDouble(gen, d_gpu, 4096, 0.0, 1.0);
    //     Destroy generator
    hiprandDestroyGenerator(gen);
    
    // Sort data    
    sort<<<1, 512>>>(d_gpu, result_gpu);
    
    hipMemcpy(data_sorted, result_gpu, 4096 * sizeof(double), hipMemcpyDeviceToHost);

    // Write the sorted data to standard out
    for (int i = 0; i < 4095; ++i){
        std::cout << data_sorted[i] << ", ";
    }
    
    std::cout << data_sorted[4095] << std::endl;
  }
