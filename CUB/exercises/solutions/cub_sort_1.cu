#include "hip/hip_runtime.h"
#include <iostream>
#include <hiprand.h>
#include <hipcub/hipcub.hpp>

template <typename T>
__global__ void sort(const T* data_in, T* data_out){
    
    typedef hipcub::BlockLoad<T, 512, 8> BlockLoadT;
    typedef hipcub::BlockRadixSort<T, 512, 8> BlockRadixSortT;
    typedef hipcub::BlockStore<T, 512, 8> BlockStoreT;
    
    __shared__ union {
        typename BlockLoadT::TempStorage load;
        typename BlockRadixSortT::TempStorage sort;
        typename BlockStoreT::TempStorage store;
    } tmp_storage;
    
    
    T items[8];
    BlockLoadT(tmp_storage.load).Load(data_in, items);
    __syncthreads();

    BlockRadixSortT(tmp_storage.sort).Sort(items);
    __syncthreads();
    
    BlockStoreT(tmp_storage.store).Store(data_out, items);
    
}

int main(){
    double* d_gpu = NULL;
    double* result_gpu = NULL;
    double* data_sorted = new double[4096];
    // Allocate memory on the GPU
    hipMalloc(&d_gpu, 4096 * sizeof(double));
    hipMalloc(&result_gpu, 4096 * sizeof(double));
    
    hiprandGenerator_t gen;
    //     Create generator
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    //     Fill array with random numbers
    hiprandGenerateNormalDouble(gen, d_gpu, 4096, 0.0, 1.0);
    //     Destroy generator
    hiprandDestroyGenerator(gen);
    
    // Sort data    
    sort<<<1, 512>>>(d_gpu, result_gpu);
    
    hipMemcpy(data_sorted, result_gpu, 4096 * sizeof(double), hipMemcpyDeviceToHost);

    // Write the sorted data to standard out
    for (int i = 0; i < 4095; ++i){
        std::cout << data_sorted[i] << ", ";
    }
    
    std::cout << data_sorted[4095] << std::endl;
}
