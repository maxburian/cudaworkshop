/** This example uses cudaManagedAllocation (available since CUDA 6.0) to allocate memory and manage
 *  the memory transfer to and from the device.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void scale(float alpha, float* a, float* c, int m){
    
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (i < m){
        c[i] = alpha * a[i];
    }
}

int main(int argc, char** argv){

    int device = 0;
    if(argc > 1) {
        device = atoi(argv[1]);
//         printf("Using device %d\n", device);
    }
    
    hipSetDevice(device);
    
    
    int m = 2048;
    float alpha = 2.0;
    float tolerance = 1e-3f;
    float* a;
    float* c;
    // TODO: Use cudaMallocManaged to allocate space for a and c
    hipMallocManaged(&a, m * sizeof(float));
    hipMallocManaged(&c, m * sizeof(float));
    for (int i = 0; i < m; ++i) a[i] = 1.0;
    // TODO: Define a 1d thread block of length 256
    dim3 blockDim(256);

    dim3 gridDim((m % 256) ? m / blockDim.x : m / blockDim.x + 1);
//     printf("gridDim(%d,%d)", gridDim.x, gridDim.y);
    
    // TODO: Call the kernel
    scale<<<gridDim, blockDim>>>(alpha, a, c, m);

    // Note: Don't forget to synchronize before you want to access the data on the host since CUDA
    //       calls are asynchronous.
    hipDeviceSynchronize();
   
    for (int i = 0; i < m; ++i){
        if (abs(c[i] - alpha * a[i]) > tolerance){
            printf("Failed! Element %d: c[%d] != %f a[%d] (%f != %f * %f)\n", i, i,alpha,i,c[i], alpha,a[i]);
            return 1;
        }
    }
    printf("Passed!\n");
    // TODO: Free memory
    hipFree(a);
    hipFree(c);
    return 0;
}
