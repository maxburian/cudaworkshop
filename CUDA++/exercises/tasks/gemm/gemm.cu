/*
 * gemm_gpu_shared.cu
 *
 * compile with: make 
 *
 * Matrices are stored as array in row-major order: 
 * A[row][col] = A[row * N + col]
 *
 * Use shared memory to speed up the matrix multiplication. We can reuse
 * the memory if we load a block of the matrix and have a thread block 
 * calculate a sub matrix.
 */


#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#define CUDA_CALL( call )\
{\
    hipError_t err = call;\
    if ( hipSuccess != err)\
        fprintf(stderr, "CUDA error for %s in %d of %s : %s.\n", #call , __LINE__ , __FILE__ \
                ,hipGetErrorString(err));\
}

// Thread block size: BLOCK_SIZE * BLOCK_SIZE
#define BLOCK_SIZE 16

// Declaration of helper functions (see bottom of file for details)
float getGflops (int, float);

/*
 *  Matrix multiplication kernel called by matrixMulOnDevice() 
 */
// TODO: Convert the kernel into a template function with typename T. Replace all occurences of 
// double with T.
__global__ void gemm_gpu_shared(double* a, double* b, double* c, int n){
    // Allocate shared memory for the two blocks aSub and bSub.
    // Use two-dimensional matrices of size BLOCK_SIZE * BLOCK_SIZE 
    __shared__ double aSub[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double bSub[BLOCK_SIZE][BLOCK_SIZE];
    
    int idxX = blockDim.x * blockIdx.x + threadIdx.x;
    int idxY = blockDim.y * blockIdx.y + threadIdx.y;
    
    // For the matrix multiplication, we need to multiply all the elements of 
    // the idxYth row of a with all the elements of the idXth column of b and 
    // sum up the results.
    double sum = 0;

    // Calculate global offset of upper left corner of thread block.
    int blockaY = blockIdx.y * BLOCK_SIZE;
    int blockbX = blockIdx.x * BLOCK_SIZE;

    for (int block = 0; block < gridDim.x; ++block){
        // Get the two sub matrices
        int blockaX = block * (BLOCK_SIZE);
        int blockbY = block * (BLOCK_SIZE);
        if (((blockaY + threadIdx.y) < n) && (blockaX + threadIdx.x) < n) {
	  aSub[threadIdx.y][threadIdx.x] = a[(blockaY + threadIdx.y) * n + blockaX + threadIdx.x];
        } else {
            aSub[threadIdx.y][threadIdx.x] = 0;
        }

        if (((blockbY + threadIdx.y) < n) && (blockbX + threadIdx.x) < n) {
            bSub[threadIdx.y][threadIdx.x] = b[(blockbY + threadIdx.y) * n + blockbX + threadIdx.x];
        } else {
            bSub[threadIdx.y][threadIdx.x] = 0;
        }
	
	__syncthreads(); // Make sure that all threads had time to read the sub matrix.
    
        if ((idxX < n) && (idxY < n))
        {
            for (int i=0; i < blockDim.x; ++i){ //assumes that we use square blocks
                sum += aSub[threadIdx.y][i] * bSub[i][threadIdx.x];
            }
        }

	__syncthreads();
	
    }
    if ((idxX < n) && (idxY < n)){    
        c[idxY * n + idxX] = sum;
    }
}

/*
 *  Matrix multiplication host function called by main() 
 */

// TODO: Convert the kernel into a template function with typename T. Replace all occurences of 
// double with T.
void matrixMulOnDevice(double* a, double* b, double* c, int n)
{
    int xGrid, yGrid;
    float time;

    // Define events for timing
    hipEvent_t start, stop;
  
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));
    
    // First calculate grid size by dividing n by BLOCK_SIZE = 16
    xGrid = (n % BLOCK_SIZE == 0) ? (n / BLOCK_SIZE) : (n / BLOCK_SIZE + 1);
    yGrid = (n % BLOCK_SIZE == 0) ? (n / BLOCK_SIZE) : (n / BLOCK_SIZE + 1);
    dim3 gridDim(xGrid, yGrid);
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);

    printf("Grid: %d, %d; block:%d, %d\n", xGrid , yGrid , BLOCK_SIZE, BLOCK_SIZE);
    
    // Invoke kernel and measure execution time 
    CUDA_CALL(hipEventRecord( start, 0 ));

    gemm_gpu_shared<<<gridDim, blockDim>>>(a, b, c, n);
    CUDA_CALL(hipGetLastError());
    CUDA_CALL(hipDeviceSynchronize()); 

    CUDA_CALL(hipEventRecord( stop, 0 ));
    CUDA_CALL(hipEventSynchronize( stop ));
    
    // Get elapsed time for kernel execution
    CUDA_CALL(hipEventElapsedTime( &time, start, stop ));
    CUDA_CALL(hipEventDestroy( start ));
    CUDA_CALL(hipEventDestroy( stop ));

    printf ("\nKernel Execution Time: %f ms (dim C: %d * %d)", time, n, n);
    printf ("\nThis corresponds to: %4.4f GFLOPS", getGflops(n, time));
  
}

int main(int argc, char** argv)
{
    int n = 1024;
    double *a, *b, *c;
    int row, col;
    double absError, maxAbsError = 0.0, sumAbsError = 0.0;
    size_t size; 

    if (argc > 1) {
       n = atoi(argv[1]);
    }

    // show banner
    printf ("\n\n     Matrix-Multiplication \n");
    printf (    "     ==========================================\n");
    printf (  "\n     Simple GEMM implemantation on GPU");  

    // echo device data
    int idevice = 0;
    CUDA_CALL(hipSetDevice(idevice));
    hipDeviceProp_t dprops;
    CUDA_CALL(hipGetDeviceProperties( &dprops, idevice ));
    printf ("\n     Device name = %s, with compute capability %d.%d \n", 
	    dprops.name, dprops.major, dprops.minor);
    printf (  "\n     Matrix size %d x %d", n, n);
  
    
    // Allocate memory for matrices (that can be accessed from host and device) 
    size = n * n * sizeof(double);
    CUDA_CALL(hipMallocManaged(&a, size));
    CUDA_CALL(hipMallocManaged(&b, size));
    CUDA_CALL(hipMallocManaged(&c, size));

    // Init matrices A and B: A = E so result will be B
    #pragma omp parallel for private(row, col)
    for (row = 0; row < n; ++row){
      for (col = 0; col < n; col++){
	a[row * n + col] = (row == col) ? 1.0 : 0.0;
	b[row * n + col] = row * n + col;
      }
    }

    // do matrix multiplication on device
    matrixMulOnDevice(a, b, c, n);
     
    // Compare results
    for ( row = 0; row < n; ++row){
      for ( col = 0; col < n; ++col) {
	
	absError = fabs ( c[row * n + col] - b[row * n + col]);
	sumAbsError += absError;
	
	if (absError > maxAbsError)
	  maxAbsError = absError;
      }
    }
    // Free memory 
    CUDA_CALL(hipFree(a));
    CUDA_CALL(hipFree(b));
    CUDA_CALL(hipFree(c));

  
    printf ("\nmaxAbsError: %4.4f, sumAbsError: %4.4f", maxAbsError, sumAbsError);
    if (maxAbsError < 2.0e-5)
      printf ("\n\nProgram terminated SUCCESSFULLY.\n\n");

    return 0;
}

/*
 *  Some helper functions
 */

// get compute performance
float getGflops (int n, float time) {
    float gf = (2.0e-6 * n * n* n / time);
    return gf;
}
