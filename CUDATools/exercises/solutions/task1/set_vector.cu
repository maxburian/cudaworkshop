
#include <hip/hip_runtime.h>
#include <cstdio>

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }



__global__ void set(const int n, float* __restrict__ const a_d, const float value) {
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	if ( i < n ) {
		a_d[ i ] = value;
	}
}


int main() {
	int n = 1024;

	CUDA_CHECK_RETURN(hipSetDevice(0));

	float *a_d = 0;
	CUDA_CHECK_RETURN(hipMalloc((void**) &a_d, n * sizeof(float)));

	float value = 3.14f;
	set<<<n/256,256>>>(n, a_d, value);
	CUDA_CHECK_RETURN(hipGetLastError());

	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	CUDA_CHECK_RETURN(hipFree(a_d));
	CUDA_CHECK_RETURN(hipDeviceReset());

	return 0;
}
