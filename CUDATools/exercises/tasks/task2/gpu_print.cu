
#include <hip/hip_runtime.h>
#include <cstdio>

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }



__global__ void print_test() {
	int i = 0;
	printf("blockIdx.x = %d, threadIdx.x = %d, i = %d\n", blockIdx.x, threadIdx.x, i);
}


int main() {
	CUDA_CHECK_RETURN(hipSetDevice(0));

	print_test<<<2,32>>>();
	CUDA_CHECK_RETURN(hipGetLastError());

	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	CUDA_CHECK_RETURN(hipDeviceReset());

	return 0;
}
