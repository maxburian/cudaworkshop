#include "hip/hip_runtime.h"
/*
 * dgemm_multigpu.cu
 *
 * compile with: nvcc -O3 -arch=sm_20 -o dgemm_multigpu dgemm_multigpu.cu -lcudart
 *
 * Matrices are stored as array in row-major order: 
 * A[row][col] = A[row * N + col]
 */

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

// Thread block size: BLOCK_SIZE * BLOCK_SIZE
#define BLOCK_SIZE 16 

#define TIMETOFLOAT(t) (1.0f*t.tv_sec + t.tv_usec/1000000.0f)

// Declaration of helper functions (see below for details)
void checkError (const char* action);
float getGflops (int, float);


/*
 *  Matrix multiplication kernel called by matrixMulOnDevice()
 */
__global__ void dgemm_gpu_simple (double *a, double *b, double *c, int n) { 

  
  double Cvalue = 0.0; 

  // Get global threadId in x and y direction
  int row = blockIdx.y * blockDim.y + threadIdx.y; 
  int col = blockIdx.x * blockDim.x + threadIdx.x; 


  // Each thread computes one element of C 
  // by accumulating results into Cvalue
  if ( row < n && col < n) {
    for (int e = 0; e < n; ++e) 
      Cvalue += a[row * n + e] * b[e * n + col]; 

    c[row * n + col] = Cvalue; 
  }
}


/*
 *  Matrix multiplication host function called by main() 
 */

void matrixMulOnDevice(const double *a, const double* b, double *c, int n) { 
  
  double *d_a0;             // matirx A in device memory GPU 0
  double *d_a1;		    // matrix A in device memory GPU 1
  double *d_b0;             // matirx B in device memory GPU 0
  double *d_b1;             // matirx B in device memory GPU 1
  double *d_c0;             // matirx C in device memory GPU 0
  double *d_c1;             // matirx C in device memory GPU 1
  size_t strip_size;
  size_t size;
  int ns;
  struct timeval start, end, total;

  // Define grid and block layout for kernel execution
  dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE); 
  dim3 gridDim( (n + BLOCK_SIZE - 1) / blockDim.x, ((n/2) + BLOCK_SIZE - 1) / blockDim.y); 
  
  size = n * n * sizeof (double);
  strip_size = (n/2) * n * sizeof (double);

  // Allocate memory for d_a0, d_b0 and d_c0 on device 0
  hipMalloc(...); //d_a0 
  checkError("allocating device memory for A in GPU 0");
  hipMalloc(...); //d_b0
  checkError("allocating device memory for B in GPU 0");
  hipMalloc(...); //d_c0
  checkError("allocating device memory for C in GPU 0");

  // Allocate memory for d_a0, d_b0 and d_c0 on device 1
  hipMalloc(...); //d_a1
  checkError("allocating device memory for A");
  hipMalloc(...); //d_b1
  checkError("allocating device memory for B");
  hipMalloc(...); //d_c1
  checkError("allocating device memory for C");

  // Copy data for a and b from host to device 0. Ensure correct device, sizes and addresses. 
  gettimeofday(&start, NULL);
  hipMemcpy(...); //d_a0
  checkError("copying data of A from host to device");
  hipMemcpy(...); //d_b0
  checkError("copying data of B from host to device");
 
  hipMemcpy(...); //d_a1
  checkError("copying data of A from host to device");
  hipMemcpy(...); //d_b1
  checkError("copying data of B from host to device");
  
  printf("Grid: %d, %d; block:%d, %d\n", gridDim.x, gridDim.y , BLOCK_SIZE, BLOCK_SIZE);
    
  // Invoke kernel in the correct devices
  // start kernel in GPU 0
  dgemm_gpu_simple<<<gridDim, blockDim>>>(d_a0, d_b0, d_c0, n); 
  // start kernel in GPU 1
  dgemm_gpu_simple<<<gridDim, blockDim>>>(d_a1, d_b1, d_c1, n);

  // Read results from device memory to C 
  hipMemcpy(...); //from d_c0
  checkError("copying results from device to host ");
  hipMemcpy(...); //from d_c1
  checkError("Copying resutls from device to host");

  gettimeofday(&end, NULL);
  timersub(&end, &start, &total);

  printf ("\nExecution Time: %f ms (dim C: %d * %d)", TIMETOFLOAT(total), n, n);

  // Free device memory. Set devices properly.
  hipFree(d_a0);
  checkError("Freeing d_a");
  hipFree(d_b0);
  checkError("Freeing d_b");
  hipFree(d_c0);
  checkError("Freeing d_c");
  hipFree(d_a1);
  checkError("Freeing d_a");
  hipFree(d_b1);
  checkError("Freeing d_b");
  hipFree(d_c1);
  checkError("Freeing d_c");


} 


/*
 *  Main program
 */
int main (int argc, char* argv[]) {

  int n = 1024; // dimension of square matrices
  double *a, *b, *c;
  int row, col;
  double absError, maxAbsError = 0.0, sumAbsError = 0.0;
  
  if (argc > 1) {
       n = atoi(argv[1]);
  }

  
  // Allocate memory for matrices on host
  assert ( a = (double*) malloc (n * n * sizeof(double)) );
  assert ( b = (double*) malloc (n * n * sizeof(double)) );
  assert ( c = (double*) malloc (n * n * sizeof(double)) );
  
  // Init matrices A and B
  #pragma omp parallel for
  for ( row = 0; row < n; row++ ) {
    for ( col = 0; col < n; col++ ) {
      a[row * n + col] = (row == col) ? 1.0 : 0.0;
      b[row * n + col] = row * n + col;
    }
  }

  // Execute matrix multiplication (on device and on host for reference
  matrixMulOnDevice (a, b, c, n);
  
  // Compare results
  for ( row = 0; row < n; ++row){
    for ( col = 0; col < n; ++col) {

      absError = fabs ( c[row * n + col] - b[row * n + col]);
      sumAbsError += absError;

      if (absError > maxAbsError)
	maxAbsError = absError;
    }
  }

  // Free memory on host
  free (a);
  free (b);
  free (c);
  
  printf ("\nmaxAbsError: %4.4f, sumAbsError: %4.4f", maxAbsError, sumAbsError);
  if (maxAbsError < 2.0e-5)
    printf ("\n\nProgram terminated SUCCESSFULLY.\n\n");

  return 0;
}



/*
 *  Some helper functions
 */

// Simple error checking function for CUDA actions
void checkError (const char* action) {
  
  hipError_t error;
  error = hipGetLastError(); 

  if (error != hipSuccess) {
    printf ("\nError while '%s': %s\nprogram terminated ...\n\n", action, hipGetErrorString(error));
    exit (EXIT_SUCCESS);
  }
}

// Compute reference results on host
void dgemm_cpu_simple (const double* a, const double* b, double* c, int n) {
  
  int row, col, k;    // loop variables
  double val;         // help variable for results
  
  /*
    PERFORM MULTIPLICATION
  */
  // loop over output rows
#pragma omp parallel for
  for ( row=0; row<n; row++ ) {
    
    // loop over output columns
    for ( col=0; col<n; col++ ) {
      
      // initialize output result to zero
      val = 0;
      
      // loop over inner dimension
      for ( k=0; k<n; k++ ) {
        // sum
        val += a[row*n+k] * b[k*n+col];
      }
      c[row*n+col] = val;
    }
  }
}

// Print the values of a matrix on the screen
// could be useful for debugging
void printMatrix (const double* m, int n) {
  
  int i, j;
  
  for (i = 0; i < n; ++i) {
    printf("\n");
    for ( j = 0; j < n; ++j) {
      printf("%6.3f", m[i * n + j]);
    }
  }
}


// get compute performance
float getGflops (int width, float time) {

	float gf = (2.0e-6 * width * width* width / time);

	return gf;
}
