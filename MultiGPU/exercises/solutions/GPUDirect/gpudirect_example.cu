#include "hip/hip_runtime.h"
/*
 * CUDA Peer to Peer Example
 */
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<sys/time.h>

#define SIZE 1048576
#define THREADS_PER_BLOCK 256
#define FLOAT(t) ((float)(t).tv_sec+((float)(t).tv_usec)/1000000)

#define CHECK_RUN( errorDescription ) { hipError_t cerror; \
        if( (cerror = hipGetLastError()) != hipSuccess ){ \
            printf("execution aborted (function : %s, file : %s, line : %d) :\n    %s -> %s\n", \
                                       __func__, __FILE__, __LINE__, \
                                       errorDescription, \
                                       hipGetErrorString(cerror) ); \
                                       return 0; } }

int run_copy_without_gpudirect(float *ha, float *tmp, float *hb, float *g0, float *g1, int size);                        
int run_copy_with_gpudirect(float *ha, float *tmp, float *hb, float *g0, float *g1, int size);
void check_results(float *a);

__global__ void kernel_gpu0 (float *g0);
__global__ void kernel_gpu1 (float *g1);

int threadsPerBlock, blocksPerGrid;

int main()
{

	struct timeval before, after, t1, t2;
	
	int size = SIZE*sizeof(float);
    	threadsPerBlock = THREADS_PER_BLOCK;
    	blocksPerGrid   = SIZE / threadsPerBlock;
	
	// Allocate memory on the host
	float *ha, *hb, *tmp;
	ha = (float*)malloc(size);
	hb = (float*)malloc(size);
	tmp = (float*)malloc(size);
	
	printf("Everything ok until here\n");	
	// Initialize the data
	int i;
	for(i=0; i<SIZE; i++)
	{
		ha[i] = 0;
		hb[i] = 0;
		tmp[i] = 0;
	}

	// Allocate memory on both the devices and enable peer access
	float *g0, *g1;
	hipSetDevice(0);
	CHECK_RUN("Set Device");
	hipSetDevice(0);
	CHECK_RUN("Set Device");
	hipDeviceEnablePeerAccess(1, 0);		// PeerGPU, flags
	CHECK_RUN("Enable Peer Access");
	hipMalloc(&g0, size);
	CHECK_RUN("Alloc g0");
	hipSetDevice(1);
	CHECK_RUN("Set Device");
	hipDeviceEnablePeerAccess(0, 0);		// PeerGPU, flags
	CHECK_RUN("Enable Peer Access");
	hipMalloc(&g1, size);
	CHECK_RUN("Alloc g1");
			
	// Copy Data Without GPUdirect
	hipSetDevice(0);
	CHECK_RUN("Set Device");
	gettimeofday(&before, NULL);
	run_copy_without_gpudirect(ha, tmp, hb, g0, g1, size);
	gettimeofday(&after, NULL);
	timersub(&after, &before, &t1);

	printf("Time without GPUdirect: %0.6f ms\n", FLOAT(t1)*1000);
	
	// Check results	
	check_results(hb);	

	// Copy Data With GPUdirect
	hipSetDevice(0);
	CHECK_RUN("Set Device");
	gettimeofday(&before, NULL);
	run_copy_with_gpudirect(ha, tmp, hb, g0, g1, size);
	gettimeofday(&after, NULL);
	timersub(&after, &before, &t2);
	
	printf("Time with GPUdirect: %0.6f ms\n", FLOAT(t2)*1000);

	// Check results
	check_results(hb);

	// Free host memory
	free(ha);
	free(hb);
	free(tmp);

	// Free memory and disable peer access
	hipSetDevice(0);
	CHECK_RUN("Set Device");
	hipDeviceDisablePeerAccess(1);
	CHECK_RUN("Disable Peer Access");
	hipFree(g0);
	CHECK_RUN("Free g0");
	hipSetDevice(1);
	CHECK_RUN("Set Device");
	hipDeviceDisablePeerAccess(0);
	CHECK_RUN("Disable Peer Access");
	hipFree(g1);
	CHECK_RUN("Free g1");
	
	return 0;

}

int run_copy_without_gpudirect(float *ha, float *tmp, float *hb, float *g0, float *g1, int size)
{
	/* TODO: Do the following here 
	 * 1. Copy ha to g0 and run kernel_gpu0 with g0
	 * 2. Modified g0 must be the input to kernel_gpu1. What do you do here?
	 * 2. Copy final result to hb
	 * Tip: Don't forget to insert hipSetDevice(0|1) at the right places
	 */
	
	// ha --> g0
	hipMemcpy(g0, ha, size, hipMemcpyDefault);
	CHECK_RUN("Memcpy ha --> g0");
 
	// Run gpu1 kernel
	kernel_gpu0<<<blocksPerGrid, threadsPerBlock>>>(g0);
	hipDeviceSynchronize();
	
	// g0 --> tmp --> g1
	hipMemcpy(tmp, g0, size, hipMemcpyDefault);
	CHECK_RUN("Memcpy g0 --> tmp");
	hipSetDevice(1);
	CHECK_RUN("Set Device");
	hipMemcpy(g1, tmp, size, hipMemcpyDefault);
	CHECK_RUN("Memcpy tmp --> g1");
	
	kernel_gpu1<<<blocksPerGrid, threadsPerBlock>>>(g1);
	//hipDeviceSynchronize();

	// g1 --> hb
	hipMemcpy(hb, g1, size, hipMemcpyDefault);
	CHECK_RUN("Memcpy g1 --> hb");

	return 0;
}


int run_copy_with_gpudirect(float *ha, float *tmp, float *hb, float *g0, float *g1, int size)
{

	/* TODO: Do the following here 
	 * 1. Copy ha to g0 and run kernel_gpu0 with g0
	 * 2. Modified g0 must be the input to kernel_gpu1. What do you do here?
	 * 2. Copy final result to hb
	 * Tip: Don't forget to insert hipSetDevice(0|1) at the right places
	 */
	 
	// ha --> g0
	hipMemcpy(g0, ha, size, hipMemcpyDefault);
	CHECK_RUN("Memcpy ha --> g0");
	 
	// Run gpu0 kernel
	kernel_gpu0<<< blocksPerGrid, threadsPerBlock >>> (g0);
	 
	// g0 --> g1
	hipMemcpy(g1, g0, size, hipMemcpyDefault);
	CHECK_RUN("Memcpy g0 -> g1");	 

	// Run gpu1 kernel
	hipSetDevice(1);
	CHECK_RUN("Set Device");
	kernel_gpu1<<< blocksPerGrid, threadsPerBlock >>> (g1);
	 
	// g1 --> hb
	hipMemcpy(hb, g1, size, hipMemcpyDefault);
	CHECK_RUN("Memcpy g1 -> hb");

	return 0;
}

__global__
void kernel_gpu0 (float *g0)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if( i < SIZE)
		if(g0[i] == 0)
			g0[i] += 1;
	
	return;

}

__global__ 
void kernel_gpu1 (float *g1)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	if(i < SIZE)
		if(g1[i] == 1)
			g1[i] += 1;
			
	return;
}

void check_results(float *hb)
{
	int i;
	for(i=0; i<SIZE; i++)
	{
		if(hb[i] != 2)
		{
			printf("Test Result Failed\n");
			return;
		}
		hb[i] = 0;
	}

	printf("Test Result Successful\n");
}