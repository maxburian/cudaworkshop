#include "hip/hip_runtime.h"
/*
 * Cuda_MM_shared.cu
 *
 * compile with: make Cuda_MM_shared_uvm
 *
 * Matrices are stored as array in row-major order: 
 * A[row][col] = A[row * N + col]
 *
 * Use shared memory to speed up the matrix multiplication. We can reuse
 * the memory if we load a block of the matrix and have a thread block 
 * calculate a sub matrix.
 */


#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

// Thread block size: BLOCK_SIZE * BLOCK_SIZE
#define BLOCK_SIZE 16

// Declaration of helper functions (see bottom of file for details)
void checkError (const char* action);
float getGflops (int, float);

/*
 *  Matrix multiplication kernel called by matrixMulOnDevice() 
 */
__global__ void dgemm_gpu_shared(double* a, double* b, double* c, int n){
    
    // Allocate shared memory for the two blocks aSub and bSub.
    // Use two-dimensional matrices of size BLOC_SIZE*BLOCK_SIZE 
    // TODO
    __shared__ double aSub[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double bSub[BLOCK_SIZE][BLOCK_SIZE];

      
    // Calculate global thread index 
    // TODO
    int idxX = blockIdx.x * blockDim.x+threadIdx.x;
    int idxY = blockIdx.y * blockDim.y+threadIdx.y;

    // For the matrix multiplication, we need to multiply all the elements of 
    // the idxYth row of a with all the elements of the idXth column of b and 
    // sum up the results.
    double sum = 0;

    // Calculate global offset of upper left corner of thread block.
    // TODO
    int blockaY = blockIdx.y * blockDim.y;
    int blockbX = blockIdx.x * blockDim.x;

    for (int block = 0; block < gridDim.x; ++block){
        // Get the two sub matrices
        int blockaX = block * (BLOCK_SIZE);
        int blockbY = block * (BLOCK_SIZE);
        if (((blockaY + threadIdx.y) < n) && (blockaX + threadIdx.x) < n) {
          // Copy block into shared memory
	  // TODO
	  asub[threadIdx.y][threadIdx.x] = a[(blockaY+threadIdx.y)*n+blockbX+threadIdx.x]; 
        } else {
            aSub[threadIdx.y][threadIdx.x] = 0;
        }

        if (((blockbY + threadIdx.y) < n) && (blockbX + threadIdx.x) < n) {
            bSub[threadIdx.y][threadIdx.x] = b[(blockbY + threadIdx.y) * n + blockbX + threadIdx.x];
        } else {
            bSub[threadIdx.y][threadIdx.x] = 0;
        }
	
	// TODO: Synchronize threads to make sure all threads are done copying
	__syncthreads();

        if ((idxX < n) && (idxY < n))
        {
            for (int i=0; i < blockDim.x; ++i){ //assumes that we use square blocks
                sum += aSub[threadIdx.y][i] * bSub[i][threadIdx.x];
            }
        }
	// TODO: Synchronize threads to make sure all threads are done with the data
	__syncthreads();	

    }
    if ((idxX < n) && (idxY < n)){    
        c[idxY * n + idxX] = sum;
    }
}



/*
 *  Matrix multiplication host function called by main() 
 */

float matrixMulOnDevice(double* a, double* b, double* c, int n)
{
    int xGrid, yGrid;
    float time;

    // Define events for timing
    hipEvent_t start, stop;
  
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // First calculate grid size by dividing n by BLOCK_SIZE = 16
    xGrid = (n % BLOCK_SIZE == 0) ? (n / BLOCK_SIZE) : (n / BLOCK_SIZE + 1);
    yGrid = (n % BLOCK_SIZE == 0) ? (n / BLOCK_SIZE) : (n / BLOCK_SIZE + 1);
    dim3 gridDim(xGrid, yGrid);
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);

    printf("\nGrid: %d, %d; block:%d, %d\n", xGrid , yGrid , BLOCK_SIZE, BLOCK_SIZE);
    
    // Invoke kernel and measure execution time 
    hipEventRecord( start, 0 );
    

    // Call the kernel 
    // TODO
    dgemm_gpu_shared<<<gridDim, blockDim>>>();
    hipDeviceSynchronize(); 

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    checkError("executing Kernel");
    
    // Get elapsed time for kernel execution
    hipEventElapsedTime( &time, start, stop );
    hipEventDestroy( start );
    hipEventDestroy( stop );

    return time;

}

int main(int argc, char** argv)
{
    int n = 1024;
    double *a, *b, *c;
    int row, col;
    double absError, maxAbsError = 0.0, sumAbsError = 0.0;
    size_t size; 
    float time;

    if (argc > 1) {
       n = atoi(argv[1]);
    }

    // show banner
    printf ("\n\nMatrix-Multiplication \n");
    printf (    "==========================================\n");
    printf (  "\nGPU implemantation using shared memory");  

    // echo device data
    int idevice = 0;
    hipSetDevice(idevice);
    hipDeviceProp_t dprops;
    hipGetDeviceProperties( &dprops, idevice );
    printf ("\nDevice name = %s, with compute capability %d.%d \n", 
	    dprops.name, dprops.major, dprops.minor);
    printf (  "\nMatrix size %d x %d", n, n);
  
    
    // Allocate memory for matrices (that can be accessed from host and device) 
    size = n * n * sizeof(double);
    ...
    checkError("hipMallocManaged: a");
    ...
    checkError("hipMallocManaged: b");  
    ...
    checkError("hipMallocManaged: c");

    // Init matrices A and B: A = E so result will be B
    #pragma omp parallel for private(row, col)
    for (row = 0; row < n; ++row){
      for (col = 0; col < n; col++){
	a[row * n + col] = (row == col) ? 1.0 : 0.0;
	b[row * n + col] = row * n + col;
      }
    }

    // do matrix multiplication on device
    time = matrixMulOnDevice(a, b, c, n);
     
    // Compare results
    for ( row = 0; row < n; ++row){
      for ( col = 0; col < n; ++col) {
	
	absError = fabs ( c[row * n + col] - b[row * n + col]);
	sumAbsError += absError;
	
	if (absError > maxAbsError)
	  maxAbsError = absError;
      }
    }
    // Free memory 
    hipFree(a);
    checkError("Freeing a");
    hipFree(b);
    checkError("Freeing b");
    hipFree(c);
    checkError("Freeing c");

  
    printf ("\nmaxAbsError: %4.4f, sumAbsError: %4.4f", maxAbsError, sumAbsError);
    if (maxAbsError < 2.0e-5) {
      printf ("\n\nProgram terminated SUCCESSFULLY.\n\n");
      printf ("\nKernel Execution Time: %f ms (dim C: %d * %d)", time, n, n);
      printf ("\nThis corresponds to: %4.4f GFLOPS\n\n", getGflops(n, time));
    } else {
      printf ("\n--> Result not correct:  check your code\n\n");
    }
    return 0;
}

/*
 *  Some helper functions
 */

// get compute performance
float getGflops (int n, float time) {

	float gf = (2.0e-6 * n * n* n / time);

	return gf;
}

// Simple error checking function for CUDA actions

void checkError (const char* action) {
  
  hipError_t error;
  error = hipGetLastError(); 

  if (error != hipSuccess) {
    printf ("\nError while '%s': %s\nprogram terminated ...\n\n", action, hipGetErrorString(error));
    exit (EXIT_FAILURE);
  }
}
