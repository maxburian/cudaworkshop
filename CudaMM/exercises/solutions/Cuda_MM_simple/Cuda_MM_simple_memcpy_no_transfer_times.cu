/*
 * Cuda_MM_simple_memcpy_no_transfer_times.cu
 *
 * compile with: make Cuda_MM_simple_memcpy_no_transfer_times
 * run with:     make runGPUmemcpyNoTransfers
 *
 * Matrices are stored as array in row-major order: 
 * A[row][col] = A[row * N + col]
 */

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>

// Thread block size: BLOCK_SIZE * BLOCK_SIZE
#define BLOCK_SIZE 16 


// Declaration of helper functions (see below for details)
void checkError (const char* action);
float getGflops (int, float);


/*
 *  Matrix multiplication kernel called by matrixMulOnDevice()
 */
__global__ void dgemm_gpu_simple (const double* __restrict__ a, const double*  __restrict__ b, double *c, int n) { 

  
  double Cvalue = 0.0; 

  // Get global threadId in x and y direction
  int row = blockIdx.y * blockDim.y + threadIdx.y; 
  int col = blockIdx.x * blockDim.x + threadIdx.x; 

  // Each thread computes one element of C 
  // by accumulating results into Cvalue
  if ( row < n && col < n) {
    for (int e = 0; e < n; ++e) 
      Cvalue += a[row * n + e] * b[e * n + col]; 
	
   c[row * n + col] = Cvalue; 
  }
}


/*
 *  Matrix multiplication host function called by main() 
 */

float matrixMulOnDevice(const double*  a, const double* b, double *c, int n) { 
  
  double *d_a;             // matirx A in device memory
  double *d_b;             // matirx B in device memory
  double *d_c;             // matirx C in device memory
  size_t size;
  float time;

  // Define grid and block layout for kernel execution
  dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE); 
  dim3 gridDim( (n + BLOCK_SIZE - 1) / blockDim.x, (n + BLOCK_SIZE - 1) / blockDim.y); 
  
  // Define events for timing
  hipEvent_t start, stop; 
  
  hipEventCreate(&start); 
  hipEventCreate(&stop); 

 
  size = n * n * sizeof (double);

  // Allocate memory for d_a, d_b and d_c on device
  hipMalloc((void**)&d_a, size);
  checkError("allocating device memory for A");
  hipMalloc((void**)&d_b, size);
  checkError("allocating device memory for B");
  hipMalloc((void**)&d_c, size);
  checkError("allocating device memory for C");

  
  // Copy data for a and b from host to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  checkError("copying data of A from host to device");
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  checkError("copying data of B from host to device");
  
  printf("\nExecution grid: %d, %d; block:%d, %d\n", gridDim.x, gridDim.y , BLOCK_SIZE, BLOCK_SIZE);
    
  // Invoke kernel and measure execution time 
  hipEventRecord(start, 0);

  dgemm_gpu_simple<<<gridDim, blockDim>>>(d_a, d_b, d_c, n); 
  checkError("kernel execution");

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  
  // Get elapsed time for kernel execution
  hipEventElapsedTime(&time, start, stop); 
  hipEventDestroy(start); 
  hipEventDestroy(stop);
  
  // Read restults from device memory to C 
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost); 
  checkError("copying results from device to host");
  
  // Free device memory 
  hipFree(d_a);
  checkError("Freeing d_a");
  hipFree(d_b);
  checkError("Freeing d_b");
  hipFree(d_c);
  checkError("Freeing d_c");

  return time; 
} 


/*
 *  Main program
 */
int main (int argc, char* argv[]) {

  int n = 1024; // dimension of square matrices
  double *a, *b, *c;
  int row, col;
  double absError, maxAbsError = 0.0, sumAbsError = 0.0;
  float time; 

  if (argc > 1) {
       n = atoi(argv[1]);
  }

  // show banner
  printf ("\n\nMatrix-Multiplication \n");
  printf (    "==========================================\n");
  printf (  "\nSimple implemantation on GPU");  

  // echo device data
  int idevice = 0;
  hipSetDevice(idevice);
  hipDeviceProp_t dprops;
  hipGetDeviceProperties( &dprops, idevice );
  printf ("\nDevice name = %s, with compute capability %d.%d \n", 
	  dprops.name, dprops.major, dprops.minor);
  printf (  "\nMatrix size %d x %d", n, n);
  
  // Allocate memory for matrices on host
  assert ( a = (double*) malloc (n * n * sizeof(double)) );
  assert ( b = (double*) malloc (n * n * sizeof(double)) );
  assert ( c = (double*) malloc (n * n * sizeof(double)) );
  
  // Init matrices A and B
  #pragma omp parallel for
  for ( row = 0; row < n; row++ ) {
    for ( col = 0; col < n; col++ ) {
      a[row * n + col] = (row == col) ? 1.0 : 0.0;
      b[row * n + col] = row * n + col;
    }
  }

  // Execute matrix multiplication (on device and on host for reference
  time = matrixMulOnDevice (a, b, c, n);
  
  // Compare results
  for ( row = 0; row < n; ++row){
    for ( col = 0; col < n; ++col) {

      absError = fabs ( c[row * n + col] - b[row * n + col]);
      sumAbsError += absError;

      if (absError > maxAbsError)
	maxAbsError = absError;
    }
  }

  // Free memory on host
  free (a);
  free (b);
  free (c);
  
  printf ("\nmaxAbsError: %4.4f, sumAbsError: %4.4f", maxAbsError, sumAbsError);
  if (maxAbsError < 2.0e-5) {
    printf ("\nProgram terminated SUCCESSFULLY.\n");
    printf ("\nKernel Execution Time: %f ms (dim C: %d * %d)", time, n, n);
    printf ("\nThis corresponds to: %4.4f GFLOPS\n\n", getGflops(n, time));
  } else { 
    printf ("\n--> Result not correct:  check your code\n\n");
  }
  return 0;
}



/*
 *  Some helper functions
 */

// Simple error checking function for CUDA actions
void checkError (const char* action) {
  
  hipError_t error;
  error = hipGetLastError(); 

  if (error != hipSuccess) {
    printf ("\nError while '%s': %s\nprogram terminated ...\n\n", action, hipGetErrorString(error));
    exit (EXIT_SUCCESS);
  }
}

// Compute reference results on host
void dgemm_cpu_simple (const double* a, const double* b, double* c, int n) {
  
  int row, col, k;    // loop variables
  double val;         // help variable for results
  
  /*
    PERFORM MULTIPLICATION
  */
  // loop over output rows
#pragma omp parallel for
  for ( row=0; row<n; row++ ) {
    
    // loop over output columns
    for ( col=0; col<n; col++ ) {
      
      // initialize output result to zero
      val = 0;
      
      // loop over inner dimension
      for ( k=0; k<n; k++ ) {
        // sum
        val += a[row*n+k] * b[k*n+col];
      }
      c[row*n+col] = val;
    }
  }
}

// Print the values of a matrix on the screen
// could be useful for debugging
void printMatrix (const double* m, int n) {
  
  int i, j;
  
  for (i = 0; i < n; ++i) {
    printf("\n");
    for ( j = 0; j < n; ++j) {
      printf("%6.3f", m[i * n + j]);
    }
  }
}


// get compute performance
float getGflops (int width, float time) {

	float gf = (2.0e-6 * width * width* width / time);

	return gf;
}
