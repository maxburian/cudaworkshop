/*
 * Cuda_MM_shared_memcpy.cu
 *
 * compile with: 
 *    -> nvcc -O3 -arch=sm_37 -o Cuda_MM_shared_memcpy Cuda_MM_shared_memcpy.cu
 *
 * Matrices are stored as array in row-major order: 
 * A[row][col] = A[row * N + col]
 *
 * Use shared memory to speed up the matrix multiplication. We can reuse
 * the memory if we load a block of the matrix and have a thread block 
 * calculate a sub matrix.
 */


#include <stdio.h>
#include <assert.h>
#ifdef _OPENMP
#include "omp.h"
#endif
#include <hip/hip_runtime.h>

// Thread block size: BLOCK_SIZE * BLOCK_SIZE
#define BLOCK_SIZE 16

// Declaration of helper functions (see bottom of file for details)
void checkError (const char* action);
float getGflops (int, float);

/*
 *  Matrix multiplication kernel called by matrixMulOnDevice() 
 */
__global__ void dgemm_gpu_shared(double* a, double* b, double* c, int n){
    
  //       Allocate shared memory for the two blocks aSub and bSub.
  //       Use two-dimensional matrices of size BLOCK_SIZE * BLOCK_SIZE 
  __shared__ double aSub[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ double bSub[BLOCK_SIZE][BLOCK_SIZE];

  // Calculate global thread index 
  int idxX = blockDim.x * blockIdx.x + threadIdx.x;
  int idxY = blockDim.y * blockIdx.y + threadIdx.y;

  // For the matrix multiplication, we need to multiply all the elements of 
  // the idxYth row of a with all the elements of the idXth column of b and 
  // sum up the results.
  double sum = 0;

  // Calculate global offset of upper left corner of thread block.
  int blockaY = blockIdx.y * BLOCK_SIZE;
  int blockbX = blockIdx.x * BLOCK_SIZE;

  for (int block = 0; block < gridDim.x; ++block){
    // Get the two sub matrices
    int blockaX = block * (BLOCK_SIZE);
    int blockbY = block * (BLOCK_SIZE);
    if (((blockaY + threadIdx.y) < n) && (blockaX + threadIdx.x) < n) {
      // Copy block into shared memory
      aSub[threadIdx.y][threadIdx.x] = a[(blockaY + threadIdx.y) * n + blockaX + threadIdx.x];
    } else {
      aSub[threadIdx.y][threadIdx.x] = 0;
    }

    if (((blockbY + threadIdx.y) < n) && (blockbX + threadIdx.x) < n) {
      bSub[threadIdx.y][threadIdx.x] = b[(blockbY + threadIdx.y) * n + blockbX + threadIdx.x];
    } else {
      bSub[threadIdx.y][threadIdx.x] = 0;
    }
	
    __syncthreads(); // Make sure that all threads had time to read the sub matrix.

    if ((idxX < n) && (idxY < n))
      {
	for (int i=0; i < blockDim.x; ++i){ //assumes that we use square blocks
	  sum += aSub[threadIdx.y][i] * bSub[i][threadIdx.x];
	}
      }

	
    __syncthreads(); // Make sure that all threads had time to process input data.

  }
  if ((idxX < n) && (idxY < n)){    
    c[idxY * n + idxX] = sum;
  }
}



/*
 *  Matrix multiplication host function called by main() 
 */

float matrixMulOnDevice(double* a, double* b, double* c, int n)
{
  int size = n * n * sizeof(double);
  int xGrid, yGrid;
  double *d_a, *d_b, *d_c;
  float time;

  // Define events for timing
  hipEvent_t start, stop;
  
  hipEventCreate(&start);
  hipEventCreate(&stop);
    
  
  // Allocate memory for d_a, d_b and d_c on device
  hipMalloc((void**) &d_a, size);
  checkError("cudaMalloc: d_a");
  hipMalloc((void**) &d_b, size);
  checkError("cudaMalloc: d_b");  
  hipMalloc((void**) &d_c, size);
  checkError("cudaMalloc: d_c");
    
    
  // First calculate grid size by dividing n by BLOCK_SIZE = 16
  xGrid = (n % BLOCK_SIZE == 0) ? (n / BLOCK_SIZE) : (n / BLOCK_SIZE + 1);
  yGrid = (n % BLOCK_SIZE == 0) ? (n / BLOCK_SIZE) : (n / BLOCK_SIZE + 1);
  dim3 gridDim(xGrid, yGrid);
  dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);

  printf("\nGrid: %d, %d; block:%d, %d\n", xGrid , yGrid , BLOCK_SIZE, BLOCK_SIZE);
    
  // Invoke kernel and measure execution time 
  hipEventRecord( start, 0 );
    
  // Copy data for a and b from host to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice); 
  checkError("copying data of A from host to device");
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice); 
  checkError("copying data of B from host to device");
    
  
  // Call the kernel 
  dgemm_gpu_shared<<<gridDim, blockDim>>>(d_a, d_b, d_c, n);

  // Read restults from device memory to C 
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
  checkError("hipMemcpyDeviceToHost");
    
  // Get elapsed time for kernel execution
  hipEventRecord( stop, 0 );
  hipEventSynchronize( stop );
  checkError("executing Kernel");
  hipEventElapsedTime( &time, start, stop );
  hipEventDestroy( start );
  hipEventDestroy( stop );

  // Free device memory 
  hipFree(d_a);
  checkError("Freeing d_a");
  hipFree(d_b);
  checkError("Freeing d_b");
  hipFree(d_c);
  checkError("Freeing d_c");

  return time;

}

int main(int argc, char** argv)
{
  int n = 1024;
  double *a, *b, *c;
  int row, col;
  double absError, maxAbsError = 0.0, sumAbsError = 0.0;
  float time; 
 
  if (argc > 1) {
    n = atoi(argv[1]);
  }

    
  // show banner
  printf ("\n\nMatrix-Multiplication \n");
  printf (    "==========================================\n");
  printf (  "\nGPU implemantation using shared memory");  

  // echo device data
  int idevice = 0;
  hipSetDevice(idevice);
  hipDeviceProp_t dprops;
  hipGetDeviceProperties( &dprops, idevice );
  printf ("\nDevice name = %s, with compute capability %d.%d \n", 
	  dprops.name, dprops.major, dprops.minor);
  printf (  "\nMatrix size %d x %d", n, n);
  
    
  // Allocate memory for matrices on host
  assert ( a = (double*) malloc (n * n * sizeof(double)) );
  assert ( b = (double*) malloc (n * n * sizeof(double)) );
  assert ( c = (double*) malloc (n * n * sizeof(double)) );
    
  // Init matrices A and B: A = E so result will be B
#pragma omp parallel for private(row, col)
  for (row = 0; row < n; ++row){
    for (col = 0; col < n; col++){
      a[row * n + col] = (row == col) ? 1.0 : 0.0;
      b[row * n + col] = row * n + col;
    }
  }

  // do matrix multiplication on device
  time = matrixMulOnDevice(a, b, c, n);
     
  // Compare results
  for ( row = 0; row < n; ++row){
    for ( col = 0; col < n; ++col) {
	
      absError = fabs ( c[row * n + col] - b[row * n + col]);
      sumAbsError += absError;
	
      if (absError > maxAbsError)
	maxAbsError = absError;
    }
  }
  // Free memory on host
  free (a);
  free (b);
  free (c);
  
  printf ("\nmaxAbsError: %4.4f, sumAbsError: %4.4f", maxAbsError, sumAbsError);
  if (maxAbsError < 2.0e-5) {
    printf ("\n\nProgram terminated SUCCESSFULLY.\n\n");
    printf ("\nKernel Execution Time: %f ms (dim C: %d * %d)", time, n, n);
    printf ("\nThis corresponds to: %4.4f GFLOPS\n\n", getGflops(n, time));
  } else {
    printf ("\n--> Result not correct:  check your code\n\n");
  }
  return 0;
}

/*
 *  Some helper functions
 */

// get compute performance
float getGflops (int n, float time) {

  float gf = (2.0e-6 * n * n* n / time);

  return gf;
}

// Simple error checking function for CUDA actions

void checkError (const char* action) {
  
  hipError_t error;
  error = hipGetLastError(); 

  if (error != hipSuccess) {
    printf ("\nError while '%s': %s\nprogram terminated ...\n\n", action, hipGetErrorString(error));
    exit (EXIT_FAILURE);
  }
}