#include "hip/hip_runtime.h"
/* Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
#include <cmath>
#include <iostream>
#include <cstdio>

#include "nvtx_macros.h"

#define CUDA_CALL( call )                                                                                          \
{                                                                                                                  \
  hipError_t err = call;                                                                                          \
  if ( hipSuccess != err)                                                                                         \
    fprintf(stderr, "CUDA error for %s in %d of %s : %s.\n", #call , __LINE__ , __FILE__ ,hipGetErrorString(err));\
}

const float PI = 2.0f*std::asin(1.0f);

__global__ void jacobi_iteration(float       * __restrict__ const a_new,
                                 float const * __restrict__ const a,
                                 const int nx,
                                 const int ny,
                                 const float weight)
{
    for(int iy  = 1+blockIdx.y*blockDim.y+threadIdx.y;
            iy  < (ny-1);
            iy += gridDim.y*blockDim.y)
    {
        for(int ix  = 1+blockIdx.x*blockDim.x+threadIdx.x;
                ix  < (nx-1);
                ix += gridDim.x*blockDim.x)
        {
            
            const float a_new_val = 0.25f* ( a[(iy+0)*nx+(ix+1)]+a[(iy+0)*nx+(ix-1)]
                                           + a[(iy+1)*nx+(ix+0)]+a[(iy-1)*nx+(ix+0)]);
            a_new[iy*nx+ix] = weight*a_new_val+(1.0f-weight)*a[iy*nx+ix];
        }
    }
}

__global__ void apply_periodic_bc(float       * __restrict__ const a,
                                  const int nx,
                                  const int ny)
{
    for(int ix  = blockIdx.x*blockDim.x+threadIdx.x;
            ix  < nx;
            ix += gridDim.x*blockDim.x)
    {
        a[     0*nx+ix]=a[(ny-2)*nx+ix];
        a[(ny-1)*nx+ix]=a[     1*nx+ix];
    }
}

void init(float * __restrict__ const a,
          float * __restrict__ const a_new,
          const int nx,
          const int ny,
          float* __restrict__ const weights,
          const int n_weights)
{
    memset(a,     0, nx*ny*sizeof(float));
    memset(a_new, 0, nx*ny*sizeof(float));
    
    // set boundary conditions
    for (int iy = 0; iy < ny; ++iy)
    {
        const float y0      = std::sin( 2.0f * PI * iy / (ny-1));
        a    [iy*nx+0]      = y0;
        a    [iy*nx+(nx-1)] = y0;
        a_new[iy*nx+0]      = y0;
        a_new[iy*nx+(nx-1)] = y0;
    }
    for (int i = 0; i < n_weights; ++i)
    {
        weights[i] = 2.0f/3.0f;
    }
}

int main()
{
    int nx = 512;
    int ny = 512;
    int n_weights = 16;
    const int iter_max = 1000;
    
    float * a;
    float * a_new;
    float * weights;

    CUDA_CALL(hipMallocManaged(&a,     nx*ny*sizeof(float)));
    CUDA_CALL(hipMallocManaged(&a_new, nx*ny*sizeof(float)));
    CUDA_CALL(hipMallocManaged(&weights, n_weights*sizeof(float)));

    init(a,a_new,nx,ny,weights,n_weights);
    
    hipEvent_t start,stop;
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));
    
    CUDA_CALL(hipDeviceSynchronize());
    CUDA_CALL(hipEventRecord(start));
    
    PUSH_RANGE("while loop",0)
    int iter = 0;

    while ( iter <= iter_max )
    {
        PUSH_RANGE("jacobi step",1)
        jacobi_iteration<<<dim3(nx/32,ny/4),dim3(32,4)>>>(a_new,a,nx,ny,weights[0]);
        CUDA_CALL(hipGetLastError());
        CUDA_CALL(hipDeviceSynchronize());
        POP_RANGE
        
        std::swap(a,a_new);
        
        PUSH_RANGE("periodic boundary conditions",2)
        //Apply periodic boundary conditions
        // TODO: launch apply_periodic_bc<<<dim3(nx/128),dim3(128)>>> kernel to apply boundary conditions on the GPU instead of CPU for loop
        for (int ix = 0; ix < nx; ++ix)
        {
            a[     0*nx+ix]=a[(ny-2)*nx+ix];
            a[(ny-1)*nx+ix]=a[     1*nx+ix];
        }

        POP_RANGE
        
        if ( 0 == iter%100 )
        {
            std::cout<<iter<<std::endl;
        }
        iter++;
    }
    
    CUDA_CALL(hipEventRecord(stop));
    CUDA_CALL(hipDeviceSynchronize());
    POP_RANGE
    
    float runtime = 0.0f;
    CUDA_CALL(hipEventElapsedTime(&runtime,start,stop));
    
    std::cout<<"Runtime "<<runtime/1000.0f<<" seconds."<<std::endl;

    hipEventDestroy(stop);
    hipEventDestroy(start);
    hipFree(weights);
    hipFree(a_new);
    hipFree(a);
    hipDeviceReset();
    
    return 0;
}

