
#include <hip/hip_runtime.h>
#include <stdio.h>

void checkError (const char* action);

int main (int argc, char** argv)
{
    int retVal = 0;    			/* return value */
    int numElements = 1<<22; 	/* 4194304 */

    float runtime = 0.0f;
    double oneSecondInMs = 1000.0;
    int oneGigaByte = 1<<30;
    hipEvent_t startEvent;
    hipEvent_t endEvent;

    float* inputBufferH;
    float* outputBufferH;
    float* inputBufferD;
    float* outputBufferD;

    hipSetDevice(0);
    checkError("Initialize CUDA device");

    hipEventCreate( &startEvent );
    checkError("Create CUDA event startEvent");
    hipEventCreate( &endEvent );
    checkError("Create CUDA event endEvent");

    //inputBufferH = (float*) malloc( numElements*sizeof(float) );
    //outputBufferH = (float*) malloc( numElements*sizeof(float) );
    hipMallocManaged( (void**) &inputBufferH, numElements*sizeof(float) );
    checkError("Allocate pinned host memory inputBufferH");
    hipMallocManaged( (void**) &outputBufferH, numElements*sizeof(float) );
    checkError("Allocate pinned host memory outputBufferH");
    hipMallocManaged( (void**) &inputBufferD, numElements*sizeof(float) );
    checkError("Allocate device memory inputBufferD");
    hipMallocManaged( (void**) &outputBufferD, numElements*sizeof(float) );
    checkError("Allocate device memory outputBufferD");

    hipEventRecord ( startEvent );
    checkError("Record CUDA event startEvent");
	
    hipMemcpy( inputBufferD, inputBufferH, numElements*sizeof(float), hipMemcpyHostToDevice );
    checkError("Copy host to device");
    hipMemcpy( outputBufferH, outputBufferD, numElements*sizeof(float), hipMemcpyDeviceToHost );
    checkError("Copy device to host");

    hipEventRecord ( endEvent );
    checkError("Record CUDA event endEvent");
    hipEventSynchronize ( endEvent );
    checkError("Synchronize CUDA event endEvent");
    hipEventElapsedTime ( &runtime, startEvent, endEvent );
    checkError("Get Elapsed Time startEvent endEvent");
	
    printf("Bidirectional Bandwidth %f (GB/s)\n", numElements*sizeof(float)*oneSecondInMs / ( oneGigaByte * runtime ) );
	
    hipFree( outputBufferD );
    hipFree( inputBufferD );
    hipFree( outputBufferH );
    hipFree( inputBufferH );
    
    hipEventDestroy( endEvent );
    hipEventDestroy( startEvent );
    hipDeviceSynchronize();
    hipDeviceReset();
    return retVal;
}

/* Simple error checking function for CUDA actions */
void checkError (const char* action) {
  hipError_t error;
  error = hipGetLastError(); 

  if (error != hipSuccess) {
    printf ("\nError while '%s': %s\nprogram terminated ...\n\n", action, hipGetErrorString(error));
    exit (EXIT_SUCCESS);
  }
}
