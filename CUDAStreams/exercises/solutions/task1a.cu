
#include <hip/hip_runtime.h>
#include <stdio.h>

void checkError (const char* action);

int main (int argc, char** argv)
{
  int retVal = 0;    		/* return value */
  int numElements = 1<<22; 	/* 4194304 */
  float runtime = 0.0f;
  double oneSecondInMs = 1000.0;
  int oneGigaByte = 1<<30;

  hipEvent_t startEvent;      /* to start timer */
  hipEvent_t endEvent;	       /* to stop timer */

  float* inputBufferH;
  float* outputBufferH;
  float* inputBufferD;
  float* outputBufferD;

  hipSetDevice(0);
  checkError("Initialize CUDA device");
  
  // create events used for timing
  hipEventCreate( &startEvent );
  checkError("Create CUDA event startEvent");
  hipEventCreate( &endEvent );
  checkError("Create CUDA event endEvent");

  // Allocate host memory
  hipHostMalloc( (void**) &inputBufferH, numElements*sizeof(float) , hipHostMallocDefault);
  checkError("Allocate pinned host memory inputBufferH");
  hipHostMalloc( (void**) &outputBufferH, numElements*sizeof(float) , hipHostMallocDefault);
  checkError("Allocate pinned host memory outputBufferH");

  // Allocate device memory
  hipMalloc( (void**) &inputBufferD, numElements*sizeof(float) );
  checkError("Allocate device memory inputBufferD");
  hipMalloc( (void**) &outputBufferD, numElements*sizeof(float) );
  checkError("Allocate device memory outputBufferD");

  // start timing
  hipEventRecord ( startEvent );
  checkError("Record CUDA event startEvent");
	
  // copy input buffer to device and output buffer to host
  hipMemcpy( inputBufferD, inputBufferH, numElements*sizeof(float), hipMemcpyHostToDevice );
  checkError("Copy host to device");
  hipMemcpy( outputBufferH, outputBufferD, numElements*sizeof(float), hipMemcpyDeviceToHost );
  checkError("Copy device to host");

  // stop timer and get elapsed runtime for data transfers
  hipEventRecord ( endEvent );
  checkError("Record CUDA event endEvent");
  hipEventSynchronize ( endEvent );
  checkError("Synchronize CUDA event endEvent");
  hipEventElapsedTime ( &runtime, startEvent, endEvent );
  checkError("Get Elapsed Time startEvent endEvent");
	
  printf("Bidirectional Bandwidth %f (GB/s)\n", numElements*sizeof(float)*oneSecondInMs / ( oneGigaByte * runtime ) );
	
  // Free host and device memory
  hipFree( outputBufferD );
  hipFree( inputBufferD );

  hipHostFree( outputBufferH );
  hipHostFree( inputBufferH );

  hipEventDestroy( endEvent );
  hipEventDestroy( startEvent );
  hipDeviceSynchronize();
  hipDeviceReset();

  return retVal;
}

/* Simple error checking function for CUDA actions */
void checkError (const char* action) {
  hipError_t error;
  error = hipGetLastError(); 

  if (error != hipSuccess) {
    printf ("\nError while '%s': %s\nprogram terminated ...\n\n", action, hipGetErrorString(error));
    exit (EXIT_SUCCESS);
  }
}
