
#include <hip/hip_runtime.h>
#include <stdio.h>

void checkError (const char* action);

int main (int argc, char** argv)
{
    int retVal = 0;    			/* return value */
	int numElements = 1<<22; 	/* 4194304 */
	float runtime = 0.0f;
	double oneSecondInMs = 1000.0;
	int oneGigaByte = 1<<30;
	hipEvent_t startEvent;
	hipEvent_t endEvent;

	hipStream_t uploadStream;
	hipStream_t downloadStream;

	float* inputBufferH;
	float* outputBufferH;
	float* inputBufferD;
	float* outputBufferD;

	hipSetDevice(0);
	checkError("Initialize CUDA device");
	hipEventCreate( &startEvent );
	checkError("Create CUDA event startEvent");
	hipEventCreate( &endEvent );
	checkError("Create CUDA event endEvent");
	hipStreamCreate( &uploadStream );
	checkError("Create CUDA stream uploadStream");
	hipStreamCreate( &downloadStream );
	checkError("Create CUDA stream downloadStream");

	hipMallocManaged( (void**) &inputBufferH, numElements*sizeof(float) );
	checkError("Allocate pinned host memory inputBufferH");
	hipMallocManaged( (void**) &outputBufferH, numElements*sizeof(float) );
	checkError("Allocate pinned host memory outputBufferH");
	hipMallocManaged( (void**) &inputBufferD, numElements*sizeof(float) );
	checkError("Allocate device memory inputBufferD");
	hipMallocManaged( (void**) &outputBufferD, numElements*sizeof(float) );
	checkError("Allocate device memory outputBufferD");

	/*event is recorded in default stream zero have timing of operations in both streams*/
	hipEventRecord ( startEvent );
	checkError("Record CUDA event startEvent");
	
	hipMemcpyAsync( inputBufferD, inputBufferH, numElements*sizeof(float), hipMemcpyHostToDevice, uploadStream );
	checkError("Start copy host to device");
	hipMemcpyAsync( outputBufferH, outputBufferD, numElements*sizeof(float), hipMemcpyDeviceToHost, downloadStream );
	checkError("Start copy device to host");

	/*synchronize default stream this will also wait for work in uploadStream and downloadStream */
	hipStreamSynchronize(0);
	hipEventRecord ( endEvent );
	checkError("Record CUDA event endEvent");
	hipEventSynchronize ( endEvent );
	checkError("Synchronize CUDA event endEvent");
	hipEventElapsedTime ( &runtime, startEvent, endEvent );
	checkError("Get Elapsed Time startEvent endEvent");
	
	printf("Bidirectional Bandwidth %f (GB/s)\n", numElements*sizeof(float)*oneSecondInMs / ( oneGigaByte * runtime ) );
	
	hipFree( outputBufferD );
	hipFree( inputBufferD );
	hipHostFree( outputBufferH );
	hipHostFree( inputBufferH );
	hipStreamDestroy( downloadStream );
	hipStreamDestroy( uploadStream );
	hipEventDestroy( endEvent );
	hipEventDestroy( startEvent );
	hipDeviceSynchronize();
	hipDeviceReset();
	return retVal;
}

/* Simple error checking function for CUDA actions */
void checkError (const char* action) {
  hipError_t error;
  error = hipGetLastError(); 

  if (error != hipSuccess) {
    printf ("\nError while '%s': %s\nprogram terminated ...\n\n", action, hipGetErrorString(error));
    exit (EXIT_SUCCESS);
  }
}
