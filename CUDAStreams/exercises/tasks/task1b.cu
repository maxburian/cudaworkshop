
#include <hip/hip_runtime.h>
#include <stdio.h>

void checkError (const char* action);

int main (int argc, char** argv)
{
    int retVal = 0;    			/* return value */
    int numElements = 1<<22; 	/* 4194304 */
    float runtime = 0.0f;
    double oneSecondInMs = 1000.0;
    int oneGigaByte = 1<<30;

    // Events to be used for timing
    hipEvent_t startEvent;
    hipEvent_t endEvent;
    
    // Streams to be created and used for parallel data transfers
    hipStream_t uploadStream;
    hipStream_t downloadStream;
    
    // Host and device pointers
    float* inputBufferH;
    float* outputBufferH;
    float* inputBufferD;
    float* outputBufferD;

    hipSetDevice(0);
    checkError("Initialize CUDA device");

    // Set up events and streams
    hipEventCreate( &startEvent );
    checkError("Create CUDA event startEvent");
    hipEventCreate( &endEvent );
    checkError("Create CUDA event endEvent");

    // TODO : Create uploadStream and downloadStream CUDA streams ...
    
    // Allocate host and device memory
    hipHostMalloc( (void**) &inputBufferH, numElements*sizeof(float) , hipHostMallocDefault);
    checkError("Allocate pinned host memory inputBufferH");
    hipHostMalloc( (void**) &outputBufferH, numElements*sizeof(float) , hipHostMallocDefault);
    checkError("Allocate pinned host memory outputBufferH");
    hipMalloc( (void**) &inputBufferD, numElements*sizeof(float) );
    checkError("Allocate device memory inputBufferD");
    hipMalloc( (void**) &outputBufferD, numElements*sizeof(float) );
    checkError("Allocate device memory outputBufferD");

    /*event is recorded in default stream zero have timing of operations in both streams*/
    hipEventRecord ( startEvent );
    checkError("Record CUDA event startEvent");
	
    //TODO: Start memcpys asynchronously in seperate streams to overlap the host to device with the device to host transfer
    hipMemcpy( inputBufferD, inputBufferH, numElements*sizeof(float), hipMemcpyHostToDevice);
    checkError("Copy host to device");
    hipMemcpy( outputBufferH, outputBufferD, numElements*sizeof(float), hipMemcpyDeviceToHost);
    checkError("Copy device to host");

    /*synchronize default stream this will also wait for work in uploadStream and downloadStream */
    hipStreamSynchronize(0);
    hipEventRecord ( endEvent );
    checkError("Record CUDA event endEvent");
    hipEventSynchronize ( endEvent );
    checkError("Synchronize CUDA event endEvent");
    hipEventElapsedTime ( &runtime, startEvent, endEvent );
    checkError("Get Elapsed Time startEvent endEvent");
	
    printf("Bidirectional Bandwidth %f (GB/s)\n", numElements*sizeof(float)*oneSecondInMs / ( oneGigaByte * runtime ) );
	
    // free host and device memory
    hipFree( outputBufferD );
    hipFree( inputBufferD );
    hipHostFree( outputBufferH );
    hipHostFree( inputBufferH );
    
    //TODO Destroy the user created streams  ...

    hipEventDestroy( endEvent );
    hipEventDestroy( startEvent );
    hipDeviceSynchronize();
    hipDeviceReset();
    
    return retVal;
}

/* Simple error checking function for CUDA actions */
void checkError (const char* action) {
  hipError_t error;
  error = hipGetLastError(); 

  if (error != hipSuccess) {
    printf ("\nError while '%s': %s\nprogram terminated ...\n\n", action, hipGetErrorString(error));
    exit (EXIT_SUCCESS);
  }
}
