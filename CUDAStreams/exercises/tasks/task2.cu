
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "hipblas.h"

void checkError (const char* action);

int main (int argc, char** argv)
{
  int retVal = 0;    			/* return value */
  int numElements = 1<<22; 	/* 4194304 */
  float alpha = 5.0f;
  float runtime = 0.0f;
  double oneSecondInMs = 1000.0;
  int oneGigaByte = 1<<30;
  int i;
  float absError, maxAbsError = 0.0, sumAbsError = 0.0;

  hipEvent_t startEvent;
  hipEvent_t endEvent;

  hipStream_t stream;
  hipblasStatus_t cuBLASstat;
  hipblasHandle_t cuBLAShandle;

  float* xH;
  float* yH;
  float* yDeviceResultH; // needed, since host will store its result to yH
  float* xD;
  float* yD;
  
  hipSetDevice(0);
  checkError("Initialize CUDA device");

  // Set up events and stream to be used
  hipEventCreate( &startEvent );
  checkError("Create CUDA event startEvent");
  hipEventCreate( &endEvent );
  checkError("Create CUDA event endEvent");
  hipStreamCreate( &stream );
  checkError("Create CUDA stream");
	
  // set up cuBLAS handle
  cuBLASstat = hipblasCreate(&cuBLAShandle);
  if (cuBLASstat != HIPBLAS_STATUS_SUCCESS) {
    printf ("CUBLAS initialization failed\n");
    return EXIT_FAILURE;
  }
	
  // allocate (pinned) host and device memory 
  hipHostMalloc( (void**) &xH, numElements*sizeof(float) , hipHostMallocDefault);
  checkError("Allocate pinned host memory xH");
  hipHostMalloc( (void**) &yH, numElements*sizeof(float) , hipHostMallocDefault);
  checkError("Allocate pinned host memory yH");
  hipHostMalloc( (void**) &yDeviceResultH, numElements*sizeof(float) , hipHostMallocDefault);
  checkError("Allocate pinned host memory yDeviceResultH");
  hipMalloc( (void**) &xD, numElements*sizeof(float) );
  checkError("Allocate device memory xD");
  hipMalloc( (void**) &yD, numElements*sizeof(float) );
  checkError("Allocate device memory yD");
	
  //Initilaize host data
  srand(124);
#pragma omp parallel for
  for (i=0; i<numElements; ++i)
    {
      xH[i] = rand() / (float)RAND_MAX;
      yH[i] = rand() / (float)RAND_MAX;
    }

  //copy input to device
  hipMemcpyAsync( xD, xH, numElements*sizeof(float), hipMemcpyHostToDevice, stream );
  checkError("Start copy host to device for x");
  hipMemcpyAsync( yD, yH, numElements*sizeof(float), hipMemcpyHostToDevice, stream );
  checkError("Start copy host to device for y");
	
  //TODO: Set cuBLAS execution stream
  //cudaStream_t stream;
  hipStreamCreate( &stream);
  hipEventCreate(&startEvent);
  
  hipEventRecord ( startEvent, stream );
  checkError("Record CUDA event startEvent");

  //TODO: Call cuBLAS SAXPY
  hipblasSaxpy(cuBLAShandle, numElements, &alpha, xD,1,yD,1);
  hipEventRecord ( endEvent, stream );
  checkError("Record CUDA event endEvent");
	
  //copy output to host
  hipMemcpyAsync( yDeviceResultH, yD, numElements*sizeof(float), hipMemcpyDeviceToHost, stream );
  checkError("Start copy device to host y");
	
  //Need to wait for upload to finish to avoid race condition
  hipEventSynchronize ( startEvent );
 
  //TODO: move cudaStreamSynchronize after host calculation to allow overlap of device side saxpy and
  //	    host side saxpy (use time ./task2 to time runtime)
  //cudaStreamSynchronize( stream );
  //checkError("Synchronize CUDA stream");

  //Compute reference asynchronously on host
#pragma omp parallel for
  for (i=0; i<numElements; ++i)
    {
      yH[i] = alpha * xH[i] + yH[i];
    }

  //AFTER host calculation
  hipStreamSynchronize( stream );
  checkError("Synchronize CUDA stream");
	
  // Compare results
  for (i=0; i<numElements; ++i)
    {
      absError = fabs ( yH[i] - yDeviceResultH[i] );
      sumAbsError += absError;
      if (absError > maxAbsError)
	maxAbsError = absError;
    }
	
  printf("maxAbsError = %f, sumAbsError = %f\n", maxAbsError, sumAbsError);
	
  hipEventSynchronize ( endEvent );
  checkError("Synchronize CUDA event endEvent");
  hipEventElapsedTime ( &runtime, startEvent, endEvent );
  checkError("Get Elapsed Time startEvent endEvent");
	
  if ( maxAbsError > 1E-6 )
    {
      printf("ERROR: Check correctnes of the code.\n");
    }
  else
    {
      printf("SAXPY Throughput %f (GB/s)\n", 3*numElements*sizeof(float)*oneSecondInMs / ( oneGigaByte * runtime ) );
    }
	
  // free host and device memory
  hipFree( yD );
  hipFree( xD );
  hipHostFree( yDeviceResultH );
  hipHostFree( yH );
  hipHostFree( xH );

  // destroy events, handle and stream
  hipblasDestroy(cuBLAShandle);
  hipStreamDestroy( stream );
  hipEventDestroy( endEvent );
  hipEventDestroy( startEvent );
  hipDeviceSynchronize();
  hipDeviceReset();

  return retVal;
}

/* Simple error checking function for CUDA actions */
void checkError (const char* action) {
  hipError_t error;
  error = hipGetLastError(); 

  if (error != hipSuccess) {
    printf ("\nError while '%s': %s\nprogram terminated ...\n\n", action, hipGetErrorString(error));
    exit (EXIT_SUCCESS);
  }
}
