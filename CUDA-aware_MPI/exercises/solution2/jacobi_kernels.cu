/* Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime.h>
#include <cstdio>

#define CUDA_RT_CALL( call )                                                                        \
{                                                                                                   \
    hipError_t cudaStatus = call;                                                                  \
    if ( hipSuccess != cudaStatus )                                                                \
        fprintf(stderr, "ERROR: CUDA RT call \"%s\" in line %d of file %s failed with %s (%d).\n",  \
                        #call, __LINE__, __FILE__, hipGetErrorString(cudaStatus), cudaStatus);     \
}

#ifdef USE_DOUBLE
    typedef double real;
    #define MPI_REAL_TYPE MPI_DOUBLE
#else
    typedef float real;
    #define MPI_REAL_TYPE MPI_FLOAT
#endif

__global__ void initialize_boundaries(
    real* __restrict__ const a_new,
    real* __restrict__ const a,
    const real pi,
    const int nx, const int ny )
{
    for (int iy = blockIdx.x * blockDim.x + threadIdx.x; 
         iy < ny; 
         iy += blockDim.x * gridDim.x) {
        const real y0 = sin( 2.0 * pi * iy / (ny-1) );
        a[     iy*nx + 0 ]         = y0;
        a[     iy*nx + (nx-1) ] = y0;
        a_new[ iy*nx + 0 ]         = y0;
        a_new[ iy*nx + (nx-1) ] = y0;
    }
}

void launch_initialize_boundaries(
    real* __restrict__ const a_new,
    real* __restrict__ const a,
    const real pi,
    const int nx, const int ny )
{
    initialize_boundaries<<<ny/128+1,128>>>( a_new, a, pi, nx, ny );
    CUDA_RT_CALL( hipGetLastError() );
}

__global__ void jacobi_kernel(
          real* __restrict__ const a_new,
    const real* __restrict__ const a,
          real* __restrict__ const l2_norm,
    const int iy_start, const int iy_end,
    const int nx)
{
    for (int iy = blockIdx.y * blockDim.y + threadIdx.y + iy_start; 
         iy < iy_end; 
         iy += blockDim.y * gridDim.y) {
    for (int ix = blockIdx.x * blockDim.x + threadIdx.x + 1; 
         ix < (nx-1); 
         ix += blockDim.x * gridDim.x) {
        const real new_val = 0.25 * ( a[ iy * nx + ix + 1 ] + a[ iy * nx + ix - 1 ]
                                    + a[ (iy+1) * nx + ix ] + a[ (iy-1) * nx + ix ] );
        a_new[ iy * nx + ix ] = new_val;
        real residue = new_val - a[ iy * nx + ix ];
        atomicAdd( l2_norm, residue*residue );
    }}
}

void launch_jacobi_kernel(
          real* __restrict__ const a_new,
    const real* __restrict__ const a,
          real* __restrict__ const l2_norm,
    const int iy_start, const int iy_end,
    const int nx,
    hipStream_t stream)
{
    dim3 dim_block(32,4,1);
    dim3 dim_grid( nx/dim_block.x+1, (iy_end-iy_start)/dim_block.y+1, 1 );
    jacobi_kernel<<<dim_grid,dim_block,0,stream>>>( a_new, a, l2_norm, iy_start, iy_end, nx );
    CUDA_RT_CALL( hipGetLastError() );
}
